#include <stdio.h>
// CUDA runtime
#include <hipblas.h>
#include <hip/hip_runtime.h>
// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

#include "cuda_util.h"


/////////工具函数//////
void AllocDevice(Matrix& dev_a)
{
   dev_a.elements =NULL;
   int batchSize = dev_a.size;
   REAL** dev_ptr= (REAL **)malloc( batchSize * sizeof(REAL*));
   for(int i=0; i < batchSize;i++)
       checkCudaErrors(hipMalloc((void**) &(dev_ptr[i]), dev_a.height * dev_a.width * sizeof(REAL*)));

   checkCudaErrors(hipMalloc( (void**) &(dev_a.elements), batchSize * sizeof(REAL*)));
   checkCudaErrors(hipMemcpy( dev_a.elements, dev_ptr , batchSize * sizeof(REAL*), hipMemcpyHostToDevice));
   free(dev_ptr);
}
void FreeDevice(Matrix& dev_a)
{
   int batchSize = dev_a.size;
   REAL** host_ptr= (REAL **)malloc( batchSize * sizeof(REAL*));
   checkCudaErrors(hipMemcpy(host_ptr, dev_a.elements , batchSize * sizeof(REAL*), hipMemcpyDeviceToHost));

   for(int i=0; i < batchSize;i++)
        hipFree( host_ptr[i]);

   checkCudaErrors(hipFree(dev_a.elements));
   free(host_ptr);
}

void Copy2Device(Matrix& dev_a,Matrix& host_A)
{
    if(dev_a.size != host_A.size)
    {
        printf("error: can't copy!");
        return;
    }
    int batchSize = host_A.size;
    REAL** host_ptr= (REAL **)malloc( batchSize * sizeof(REAL*));
    checkCudaErrors(hipMemcpy(host_ptr, dev_a.elements , batchSize * sizeof(REAL*), hipMemcpyDeviceToHost));

    //ShowMatrixByRow2(host_A);
    for(int i=0; i < batchSize;i++)
    {
        // printf("host_ptr==%p\n ",host_ptr[i]);
        checkCudaErrors(hipMemcpy(host_ptr[i], host_A.elements[i], dev_a.height * dev_a.width*sizeof(REAL), hipMemcpyHostToDevice));
    }
    free(host_ptr);
}
void CopyBack2Host(Matrix& host_A,Matrix& dev_a)
{
    if(dev_a.size != host_A.size)
    {
        printf("error: can't copy to host!");
        return;
    }
    int batchSize = dev_a.size;
    REAL** dev_ptr= (REAL **)malloc( batchSize * sizeof(REAL*));
    checkCudaErrors(hipMemcpy(dev_ptr, dev_a.elements , batchSize * sizeof(REAL*), hipMemcpyDeviceToHost));
    for(int i=0; i < batchSize;i++)
    {
         checkCudaErrors(hipMemcpy(host_A.elements[i], dev_ptr[i], dev_a.height * dev_a.width*sizeof(REAL), hipMemcpyDeviceToHost));
    }
    free(dev_ptr);
}




